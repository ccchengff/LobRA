#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"
#include "hetu/impl/utils/offset_calculator.cuh"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void check_finite_kernel(const spec_t* input, size_t size, float* output,
                                    const OffsetCalculator* in_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  auto in_offset = in_offset_calculator->get(idx);
  if (!isfinite(float(input[in_offset])))
    output[0] = 1.f;
  else 
    output[0] = 0.f;
}

void CheckFiniteCuda(const NDArray& input, NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);

  size_t size = input->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray in_offset_calculator_arr, out_offset_calculator_arr;
  OffsetCalculator *in_offset_calculator, *out_offset_calculator;
  std::tie(in_offset_calculator_arr, in_offset_calculator) =
    AllocOffsetCalculator(input, stream);
  std::tie(out_offset_calculator_arr, out_offset_calculator) = 
    AllocOffsetCalculator(output, stream);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "CheckFiniteCuda", [&]() {
      check_finite_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), size, output->data_ptr<float>(),
        in_offset_calculator);
    });
  NDArray::MarkUsedBy({input, output, in_offset_calculator_arr,
                      out_offset_calculator_arr}, stream);
}

} // namespace impl
} // namespace hetu
